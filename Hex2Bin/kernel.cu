#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>

#define TEXT_PROLOG_SIZE 8
#define TEXT_LINE_SIZE 58
#define BYTES_PER_LINE 16

#define NBLOCK 1024

/*
 * one block per text line
 * one thread per hexadecimal bytes in line
 */
__global__ void hexConvertKernel(unsigned char* result, char* input)
{
	unsigned char d0, d1;
	int lineId = blockIdx.x;
	int byteId = threadIdx.x;
	
	d0 = input[lineId * TEXT_LINE_SIZE + TEXT_PROLOG_SIZE + byteId * 3 + 1]; // * 3 because 1 escape + 2 hex digits
	d1 = input[lineId * TEXT_LINE_SIZE + TEXT_PROLOG_SIZE + byteId * 3 + 2];

	/*
	 * convert char digits to binary value in cuda core
	 */
	if (d0 == '?') d0 = 0;
	else if (d0 <= '9') d0 -= '0';
	else if (d0 <= 'F') d0 = d0 - 'A' + 10;
	
	if (d1 == '?') d1 = 0;
	else if (d1 <= '9') d1 -= '0';
	else if (d1 <= 'F') d1 = d1 - 'A' + 10;

	result[lineId * BYTES_PER_LINE + byteId] = d0 << 4 | d1;
}

/*
 * check if digit is a hexadecimal digit (lower char not allowed)
 */
bool isHexDigit(char digit)
{
	return ((digit >= '0' && digit <= '9') || (digit >= 'A' && digit <= 'F'));
}

/*
 * convert hexadecimal digit to 4 bits integer (lower char not allowed)
 */
unsigned char hexDigit2Byte(char digit)
{
	if (digit == '?') digit = 0;
	else if (digit <= '9') digit -= '0';
	else if (digit <= 'F') digit = digit - 'A' + 10;
	return digit;
}


long getFileSize(FILE* fd)
{
	long size;
	fseek(fd, 0, SEEK_END);
	size = ftell(fd);
	fseek(fd, 0, SEEK_SET);
	return size;
}

/**
 *  convert hexdecimal content (like above) with the CPU into binary format (by removing address and convert hexadecimals digits)
 *  00401000 56 8D 44 24 08 50 8B F1 E8 1C 1B 00 00 C7 06 08
 *  00401010 BB 42 00 8B C6 5E C2 04 00 CC CC CC CC CC CC CC
 *  00401020 C7 01 08 BB 42 00 E9 26 1C 00 00 CC CC CC CC CC 
 *  usr_in : user allocated buffer with hexadecimal content
 *  input_size : size of user allocated buffer
 *  usr_out: output allocated buffer to store binary content
 */
size_t CPU_Hex2Bin(char* usr_in,size_t input_size, unsigned char* usr_out)
{
	int nRound = input_size / TEXT_LINE_SIZE;
	int remainingBytes = input_size % TEXT_LINE_SIZE;
	int nHexBytes = (remainingBytes - TEXT_PROLOG_SIZE) / 3;

	/* convert remaining lines */
	for (int i = 0; i < nRound; i++)
	{
		for (int j = 0; j < BYTES_PER_LINE; j++)
		{
			unsigned char d0 = hexDigit2Byte(usr_in[i * TEXT_LINE_SIZE + TEXT_PROLOG_SIZE + j * 3 + 1]);
			unsigned char d1 = hexDigit2Byte(usr_in[i * TEXT_LINE_SIZE + TEXT_PROLOG_SIZE + j * 3 + 2]);

			usr_out[i * BYTES_PER_LINE + j] = d0 << 4 | d1;
		}
	}

	/* convert last partial line */
	for (int i = 0; i < nHexBytes; i++)
	{
		unsigned char d0 = hexDigit2Byte(usr_in[nRound * TEXT_LINE_SIZE + TEXT_PROLOG_SIZE + i * 3 + 1]);
		unsigned char d1 = hexDigit2Byte(usr_in[nRound * TEXT_LINE_SIZE + TEXT_PROLOG_SIZE + i * 3 + 2]);

		usr_out[nRound * BYTES_PER_LINE + i] = d0 << 4 | d1;
	}

	return nRound * BYTES_PER_LINE + nHexBytes;
}


/**
 *  convert hexdecimal content (like above) with the GPU into binary format (by removing address and convert hexadecimals digits)
 *  00401000 56 8D 44 24 08 50 8B F1 E8 1C 1B 00 00 C7 06 08
 *  00401010 BB 42 00 8B C6 5E C2 04 00 CC CC CC CC CC CC CC
 *  00401020 C7 01 08 BB 42 00 E9 26 1C 00 00 CC CC CC CC CC
 *  filename_in : path to hexadecimal file
 *  filename_out : path to output file
 *  
 *  file is divided in big chunk of size (NBLOCK * TEXT_LINE_SIZE)
 *  The GPU process the big chunk with NBLOCK blocks and BYTES_PER_LINE (16) threads per block.
 *  The remaining lines is processed with the CPU.
 */
hipError_t CUDA_HexFile2Bin(char* filename_in,char* filename_out)
{
	// indicate error with GPU
	hipError_t cudaStatus;
	// input and output file descriptor
	FILE *in = NULL, *out = NULL;
	// device input buffer and user input buffer
	char* dev_in = NULL, *usr_in = NULL;
	// device output buffer and user output buffer
	unsigned char* dev_out = NULL, *usr_out = NULL;

	// open input file in read mode
	in = fopen(filename_in, "rb");

	if (in == NULL)
	{
		perror("fopen input file");
		goto Error;
	}

	// open output file in write mode
	out = fopen(filename_out, "wb");

	if (out == NULL)
	{
		perror("fopen output file");
		goto Error;
	}


	long file_size = getFileSize(in);

	printf("[i] file size : %d bytes\n", file_size);
	printf("[i] block size: %d bytes\n", NBLOCK * TEXT_LINE_SIZE);

	// divide the file into big chunk (NBLOCK * TEXT_LINE_SIZE)
	unsigned int nRound = file_size / (NBLOCK * TEXT_LINE_SIZE);
	unsigned int remainingBytes = file_size % (NBLOCK * TEXT_LINE_SIZE);

	printf("[i] nRound : %d\n", nRound);
	printf("[i] remainingBytes : %d\n", remainingBytes);

	// allocate user buffer for one big chunk
	usr_in  = (char*)malloc(NBLOCK * TEXT_LINE_SIZE);
	// allocate output buffer for one big chunk
	usr_out = (unsigned char*)malloc(NBLOCK * BYTES_PER_LINE);

	// Allocate GPU buffers for two vectors (one input, one output).
	cudaStatus = hipMalloc((void**)&dev_in, NBLOCK * TEXT_LINE_SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_out, NBLOCK * BYTES_PER_LINE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	for (int i = 0; i < nRound; i++)
	{
		fread(usr_in, TEXT_LINE_SIZE, NBLOCK, in);

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_in, usr_in, NBLOCK * TEXT_LINE_SIZE, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// Launch a kernel on the GPU with one thread for each element.
		hexConvertKernel <<<NBLOCK, BYTES_PER_LINE >> > (dev_out, dev_in);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(usr_out, dev_out, NBLOCK * BYTES_PER_LINE, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		fwrite(usr_out, BYTES_PER_LINE, NBLOCK, out);
	}

	fread(usr_in,remainingBytes,1,in);
	size_t nbytes = CPU_Hex2Bin(usr_in, remainingBytes, usr_out);
	fwrite(usr_out, nbytes, 1, out);

Error:
	hipFree(dev_in);
	hipFree(dev_out);
	free(usr_out);
	free(usr_in);
	fclose(out);
	fclose(in);

	return cudaStatus;
}

int main(int argc,char** argv)
{
	hipError_t cudaStatus;

	if (argc != 3)
	{
		printf("Usage: %s <hexadecimal input file> <output file>\n",argv[0]);
		exit(1);
	}

	cudaStatus = CUDA_HexFile2Bin(argv[1], argv[2]);
	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
